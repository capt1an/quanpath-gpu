#include "hip/hip_runtime.h"
﻿
#include ""
#include "cuquanpath.h"

int numQubits = 16;
int numThreads = 32;
int numDepths = 1000;

int numHighQubits = log2(numThreads);		  // high-order qubits
int numLowQubits = numQubits - numHighQubits; // low-order qubits
ll lenSv = (1 << numQubits);				  // the length of the local state vector

// Generate a separable quantum circuit
QCircuit separableqc()
{
	QCircuit qc(numQubits, "separable");
	qc.numLowQubits = numLowQubits;
	qc.numHighQubits = numHighQubits;
	for (int layer = 0; layer < numDepths; layer++)
	{
		if (layer % 5 == 0)
		{
			for (int i = numQubits - 1; i > numLowQubits; i -= 2)
			{
				qc.h(i);
			}
			for (int i = 1; i < numLowQubits; i += 2)
			{
				qc.cy(i, i - 1);
			}
		}
		else if (layer % 5 == 1)
		{
			for (int i = 0; i < numQubits; i++)
			{
				if (i % 3 == 0)
					qc.rx((double)(i + 1) / numQubits, i);
				if (i % 3 == 1)
					qc.ry((double)(i + 1) / numQubits, i);
				if (i % 3 == 2)
					qc.rz((double)(i + 1) / numQubits, i);
			}
		}
		else if (layer % 5 == 2)
		{

			for (int i = 0; i < numQubits; i++)
			{
				if (i % 3 == 0)
					qc.x(i);
				if (i % 3 == 1)
					qc.y(i);
				if (i % 3 == 2)
					qc.z(i);
			}
		}
		else if (layer % 5 == 3)
		{

			for (int i = 0; i < numQubits; i++)
			{
				if (i >= numLowQubits) // 高阶部分
				{
					qc.rx((double)(layer + 1) / 200.0, i);
				}
				else // 低阶部分
				{
					qc.ry((double)(layer + 1) / 200.0, i);
				}
			}
		}
		else
		{
			for (int i = numQubits - 1; i > numLowQubits; i -= 2)
			{
				qc.h(i);
			}
			for (int i = 1; i < numLowQubits; i += 2)
			{
				qc.cz(i, i - 1);
			}
		}
		if(layer != numDepths - 1)
			qc.barrier();
	}

	return qc;
}

int main()
{
	// Initialize a quantum circuit
	QCircuit qc = separableqc();

	// 模拟次数
	int numSimulations = 5;
	// 存储模拟时间的向量
	vector<double> simulationTimes;

	for (int times = 0; times < numSimulations; ++times)
	{
		// 获取开始时间
		auto start = chrono::high_resolution_clock::now();
		QuanPath(qc);

		// 获取结束时间
		auto end = chrono::high_resolution_clock::now();

		chrono::duration<double> duration = end - start;
		simulationTimes.push_back(duration.count());
		cout << "Simulation " << times + 1 << " completed in " << duration.count() << " seconds." << endl;
	}
	double average = accumulate(simulationTimes.begin(), simulationTimes.end(), 0.0) / simulationTimes.size();
	auto minIt = min_element(simulationTimes.begin(), simulationTimes.end());
	cout << "Average: " << average << std::endl;
	cout << "Min:" << *minIt << std::endl;
	return 0;
}