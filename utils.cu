
#include "utils.h"


CudaUtils::CudaUtils(){

    hipEventCreate(&start);
    hipEventCreate(&stop);

}

hipError_t CudaUtils::startTiming(const std::string& taskName) {
    
    currentTaskName = taskName;
    return HANDLE_CUDA_ERROR(hipEventRecord(start, 0));
}

hipError_t CudaUtils::stopTiming()
{
    HANDLE_CUDA_ERROR(hipEventRecord(stop, 0));
    HANDLE_CUDA_ERROR(hipEventSynchronize(stop));

    float elapsedTime = 0.0f;
    HANDLE_CUDA_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    if (currentTaskName.empty())
        std::cout << "[CUDA] Kernel execution time: " << elapsedTime << " ms" << std::endl;
    else
        std::cout << "[CUDA] " << currentTaskName << " execution time: " << elapsedTime << " ms" << std::endl;

    return hipSuccess;
}


void CudaUtils::writeDeviceStateVectorToFile(DTYPE *deviceSv, long long stateVectorLen, const char *filename)
{
    // 分配主机内存
    DTYPE *hostSv = (DTYPE *)malloc(stateVectorLen * sizeof(DTYPE));
    if (hostSv == NULL)
    {
        fprintf(stderr, "Host memory allocation failed!\n");
        exit(EXIT_FAILURE);
    }

    // 拷贝 deviceSv 到主机
    HANDLE_CUDA_ERROR(hipMemcpy(hostSv, deviceSv, stateVectorLen * sizeof(DTYPE), hipMemcpyDeviceToHost));

    // 打开文件
    FILE *outFile = fopen(filename, "w");
    if (outFile == NULL)
    {
        fprintf(stderr, "Failed to open output file: %s\n", filename);
        free(hostSv);
        exit(EXIT_FAILURE);
    }

    // 写入数据
    for (long long i = 0; i < stateVectorLen; ++i)
    {
        DTYPE val = hostSv[i];
        fprintf(outFile, "%.7f %.7f\n", hipCreal(val), hipCimag(val));
    }

    fclose(outFile);
    printf("State vector written to %s\n", filename);

    // 释放主机内存
    free(hostSv);
}

CudaUtils::~CudaUtils(){
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}
